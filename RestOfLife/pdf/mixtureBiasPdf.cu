#include "hip/hip_runtime.h"

#include "pdf.cuh"
#include "../shaders/sysparameter.h"
#include "../shaders/FunctionIdx.h"
#include <cuda/random.h>
//rtDeclareVariable(rtCallableProgramId<float(pdf_in&)>, p0_value, , );
//rtDeclareVariable(rtCallableProgramId<float(pdf_in&)>, p1_value, , );

//rtDeclareVariable(float, bias, , );
extern "C" __constant__ SysParamter Parameter;

extern "C" __device__ float __direct_callable__mixtureBIAS_value(pdf_in &in) {
    pdfCallfun* p0 = (pdfCallfun*)Parameter.pdf.p0;
    int p0_value = p0->pdfValIdx + NUM_CALLABLE_CAMERA +
        NUM_CALLABE_TEX_IDS + NUM_CALLABLE_MAT_IDS;
    pdfCallfun* p1 = (pdfCallfun*)Parameter.pdf.p1;
    int p1_value = p1->pdfValIdx + NUM_CALLABLE_CAMERA +
        NUM_CALLABE_TEX_IDS + NUM_CALLABLE_MAT_IDS;
    float val0= optixDirectCall<float, pdf_in&>(p0_value, in);
    float val1 = optixDirectCall<float, pdf_in&>(p1_value, in);
    return 0.5f * val0 + 0.5f * val1;

}

//rtDeclareVariable(rtCallableProgramId<float3(pdf_in&, uint32_t&)>, p0_generate, , );
//rtDeclareVariable(rtCallableProgramId<float3(pdf_in&, uint32_t&)>, p1_generate, , );

extern "C" __device__ float3 __direct_callable__mixtureBIAS_generate(pdf_in &in, uint32_t& seed) {
    pdfCallfun* p0= (pdfCallfun*)Parameter.pdf.p0;
    int p0_generate = p0->pdfGenIdx + NUM_CALLABLE_CAMERA +
        NUM_CALLABE_TEX_IDS + NUM_CALLABLE_MAT_IDS;
    pdfCallfun* p1 = (pdfCallfun*)Parameter.pdf.p1;
    int p1_generate = p1->pdfGenIdx + NUM_CALLABLE_CAMERA +
        NUM_CALLABE_TEX_IDS + NUM_CALLABLE_MAT_IDS;
    if (rnd(seed) < Parameter.pdf.bias)
        return optixDirectCall<float3,pdf_in&,uint32_t&>( p0_generate, in, seed);
    else
        return optixDirectCall<float3,pdf_in&,uint32_t&>( p1_generate, in, seed);
}
